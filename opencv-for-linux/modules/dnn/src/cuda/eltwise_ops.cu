#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "functors.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "vector_traits.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

namespace raw {
    template <class T, class Functor, std::size_t N, class ...FunctorArgs>
    __global__ void eltwise_op_vec(Span<T> output, View<T> x, View<T> y, FunctorArgs ...functorArgs) {
        using vector_type = get_vector_type_t<T, N>;

        auto output_vPtr = vector_type::get_pointer(output.data());
        auto x_vPtr = vector_type::get_pointer(x.data());
        auto y_vPtr = vector_type::get_pointer(y.data());

        Functor functor(functorArgs...);

        for (auto i : grid_stride_range(output.size() / vector_type::size())) {
            vector_type vec_x, vec_y;
            v_load(vec_x, x_vPtr[i]);
            v_load(vec_y, y_vPtr[i]);
            for (int j = 0; j < vector_type::size(); j++)
                vec_x.data[j] = functor(vec_x.data[j], vec_y.data[j]);
            v_store(output_vPtr[i], vec_x);
        }
    }
}

template <class T, template <class> class EltwiseOp, std::size_t N, class ...EltwiseOpArgs> static
void launch_vectorized_eltwise_op(const Stream& stream, Span<T> output, View<T> x, View<T> y, EltwiseOpArgs ...eltwiseOpArgs) {
    CV_Assert(x.size() == y.size());
    CV_Assert(x.size() == output.size());
    CV_Assert(is_fully_aligned<T>(output, N));
    CV_Assert(is_fully_aligned<T>(x, N));
    CV_Assert(is_fully_aligned<T>(y, N));

    auto kernel = raw::eltwise_op_vec<T, EltwiseOp<T>, N, EltwiseOpArgs...>;
    auto policy = make_policy(kernel, output.size() / N, 0, stream);
    launch_kernel(kernel, policy, output, x, y, eltwiseOpArgs...);
}

template <class T, template <class> class EltwiseOp, class ...EltwiseOpArgs> static
void eltwise_op(const Stream& stream, Span<T> output, View<T> x, View<T> y, EltwiseOpArgs ...eltwiseOpArgs) {
    CV_Assert(x.size() == y.size());
    CV_Assert(x.size() == output.size());

    if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(x, 4) && is_fully_aligned<T>(y, 4)) {
        launch_vectorized_eltwise_op<T, EltwiseOp, 4>(stream, output, x, y, eltwiseOpArgs...);
    } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(x, 2) && is_fully_aligned<T>(y, 2)) {
        launch_vectorized_eltwise_op<T, EltwiseOp, 2>(stream, output, x, y, eltwiseOpArgs...);
    } else {
        launch_vectorized_eltwise_op<T, EltwiseOp, 1>(stream, output, x, y, eltwiseOpArgs...);
    }
}

template <class T>
void eltwise_max_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op<T, max_functor>(stream, output, x, y);
}

template <class T>
void eltwise_sum_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op<T, sum_functor>(stream, output, x, y);
}

template <class T>
void eltwise_sum_coeff_2(const Stream& stream, Span<T> output, T coeff_x, View<T> x, T coeff_y, View<T> y) {
    eltwise_op<T, scaled_sum_functor>(stream, output, x, y, coeff_x, coeff_y);
}

template <class T>
void eltwise_prod_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op<T, product_functor>(stream, output, x, y);
}

template <class T>
void eltwise_div_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
    eltwise_op<T, div_functor>(stream, output, x, y);
}

#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 530)
    template void eltwise_div_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
    template void eltwise_prod_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
    template void eltwise_sum_coeff_2(const Stream&, Span<__half>, __half, View<__half>, __half, View<__half>);
    template void eltwise_sum_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
    template void eltwise_max_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
#endif
    template void eltwise_div_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);
    template void eltwise_prod_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);
    template void eltwise_sum_coeff_2(const Stream&, Span<float>, float, View<float>, float, View<float>);
    template void eltwise_sum_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);
    template void eltwise_max_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
